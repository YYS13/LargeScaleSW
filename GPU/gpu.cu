#include "hip/hip_runtime.h"
#include "functions.cuh"


int main(){
    clock_t start, end;
    // 設定由哪個 gpu 執行
    int device_id = 0;
    ErrorCheck(hipSetDevice(device_id), __FILE__, __LINE__);

    hipDeviceProp_t prop;
    ErrorCheck(hipGetDeviceProperties(&prop, device_id), __FILE__, __LINE__);
    printf("GPU global memory space : %zu Bytes\n", prop.totalGlobalMem);
    // 讀取數據
    char *nDNA = read_from_file("../data/nDNA.txt");
    char *mtDNA = read_from_file("../data/mtDNA.txt");
    printf("nDNA length: %lld\nmtDNA length : %d\n", (long long)strlen(nDNA), (int)strlen(mtDNA));

    //設定 panalty 參數
    int *panalty_score = (int *)malloc(sizeof(int) * 4);
    for(int i = 0; i < 4; i++){
        switch (i)
        {
        case 0:
            panalty_score[i] = MATCH;   
            break;
        case 1:
            panalty_score[i] = MISMATCH;
            break;
        case 2:
            panalty_score[i] = EXTEND_GAP;
            break;
        case 3:
            panalty_score[i] = OPEN_GAP;
            break;
        default:
            break;
        }
    }

    //分配 GPU 空間
    hipMemcpyToSymbol(HIP_SYMBOL(panalty), panalty_score, 4 * sizeof(int));
    int *F;
    ErrorCheck(hipMalloc((int**) &F, sizeof(int) * strlen(mtDNA)), __FILE__, __LINE__);
    int *E;
    ErrorCheck(hipMalloc((int**) &E, sizeof(int) * 6400), __FILE__, __LINE__);
    int *H;
    ErrorCheck(hipMalloc((int**)&H, sizeof(int) * (strlen(mtDNA) + 1) * 6401), __FILE__, __LINE__);
    int *global_max_score;
    ErrorCheck(hipMalloc((int**)&global_max_score, sizeof(int)), __FILE__, __LINE__);
    int *global_max_i;
    ErrorCheck(hipMalloc((int**)&global_max_i, sizeof(int)), __FILE__, __LINE__);
    int *global_max_j;
    ErrorCheck(hipMalloc((int**)&global_max_j, sizeof(int)), __FILE__, __LINE__);
    // copy mtDNA 到 constant memory
    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(device_mtDNA), mtDNA, strlen(mtDNA) + 1, 0, hipMemcpyHostToDevice), __FILE__, __LINE__);

    //fill F vector with -∞
    int threadsPerBlock = 512;
    int blocksPerGrid = (strlen(mtDNA) + threadsPerBlock - 1) / threadsPerBlock;
    fill_array_value<<<blocksPerGrid, threadsPerBlock>>>(F, INT_MIN - OPEN_GAP, strlen(mtDNA));
    hipDeviceSynchronize();

    // fill first row and col with 0 in H
    start = clock();
    int maxLen = MAX(strlen(mtDNA), 6400);
    threadsPerBlock = 512;
    blocksPerGrid = (maxLen + threadsPerBlock - 1) / threadsPerBlock;
    initializeH<<<blocksPerGrid, threadsPerBlock>>>(H, strlen(mtDNA), 6400);
    hipDeviceSynchronize();
    end = clock();
    double elapsed_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("填充 H 花 %.6f 秒\n", elapsed_time);

    // proint_arrInfo<<<1, 1>>>(H, 6587);
    // hipDeviceSynchronize();


    //根據nDNA長度切段去執行每個subMatrix
    int epoch = (int)(strlen(nDNA) / (size_t)6400);
    threadsPerBlock = 263;
    blocksPerGrid = 40;
    for(int i = 0; i < epoch; i++){
        printf("%d / %d \n", i+1, epoch);
        char *slice = substring(nDNA, 6400 * i, 6400);
        //copy nDNA slice to constant memory
        ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(device_slice_nDNA), slice, strlen(slice) + 1, 0, hipMemcpyHostToDevice), __FILE__, __LINE__);
        // fill E vector with -∞
        blocksPerGrid = (6400 + threadsPerBlock - 1) / threadsPerBlock;
        fill_array_value<<<blocksPerGrid, threadsPerBlock>>>(E, INT_MIN - OPEN_GAP, strlen(slice));
        //start caculate submatrix
        int outer_diag = blocksPerGrid + (strlen(mtDNA) / threadsPerBlock);
        for(int i = 0; i < outer_diag; i++){
            // do first part in all blocks
            cal_first_phase<<<blocksPerGrid, threadsPerBlock>>>(i, threadsPerBlock, (int)(strlen(slice) / blocksPerGrid), strlen(slice), strlen(mtDNA), E, F, H, global_max_score, global_max_i, global_max_j);
            hipDeviceSynchronize();
            // do second part in all blocks
            cal_second_phase<<<blocksPerGrid, threadsPerBlock>>>(i, threadsPerBlock, (int)(strlen(slice) / blocksPerGrid), strlen(slice), strlen(mtDNA), E, F, H, global_max_score, global_max_i, global_max_j);
            hipDeviceSynchronize();
        }
        printf("max score : %d at (%d, %d)\n", *global_max_score, *global_max_i, *global_max_j);
    }


    //free memory space
    hipFree(F);
    hipFree(E);
    hipFree(H);
    free(mtDNA); 
    free(nDNA);

    return 0;
}